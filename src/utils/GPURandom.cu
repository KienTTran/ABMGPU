#include "hip/hip_runtime.h"
//
// Created by kient on 6/17/2023.
//

#include "GPURandom.cuh"
#include "../utils/GPUUtils.cuh"

GPURandom::GPURandom() {
    d_states = nullptr;
    n_threads = 1024;
}

GPURandom::~GPURandom() {
    hipFree(d_states);
}

__global__ void setup(hiprandState *state)
{
    auto id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(123456, id, 0, &state[id]);
}

void GPURandom::init(int n) {
    hipMalloc((void **) &d_states, sizeof(hiprandState) * n);
    n_blocks = (n + n_threads + 1) / n_threads;
    setup<<<n_blocks,n_threads>>>(d_states);
    checkCudaErr(hipDeviceSynchronize());
    checkCudaErr(hipPeekAtLastError());
}

void GPURandom::free() {
    hipFree(d_states);
}

